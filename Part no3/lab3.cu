#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius = 16;

#define FILTER_LENGTH 	(2 * 16 + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.05 
#define TYPE float
#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

 //GPU code
 
 
 __constant__ TYPE d_Filter[FILTER_LENGTH];  

__global__ void convolROWS(TYPE* d_Buffer, 
		       TYPE* d_Input, 
		       int imageH, 
		       int imageW, 
		       int filterR)
{

      int k;
      int blockId = blockIdx.y * gridDim.x + blockIdx.x;
      int tx = threadIdx.y * blockDim.x + threadIdx.x;
      int threadId = blockId * (blockDim.x * blockDim.y) + tx;
      TYPE sum = 0;
      //tile = threadBlock = 1024
      __shared__ TYPE s_input[1024];
      
      //load to shared memory

      s_input[tx] = d_Input[threadId];
      __syncthreads();  // wait for all shared data to load
      
      //result
      for (k = -filterR; k <= filterR; k++) {
	int d = threadIdx.x + k;
	
	if (d >= 0 && (d < imageW)) {
          sum += s_input[tx + k] * d_Filter[filterR - k];
        }     

        d_Buffer[threadId] = sum;	
      }
      
      
      
}

__global__ void convolCOL(TYPE* d_Output_GPU, 
		       TYPE* d_Buffer, 
		       int imageH, 
		       int imageW, 
		       int filterR)
{

      int k;
      int blockId = blockIdx.y * gridDim.x + blockIdx.x;
      int ty = threadIdx.x * blockDim.y + threadIdx.y;

      TYPE sum = 0;
      //tile = threadBlock = 1024
      __shared__ TYPE s_input[1024];
      
    // load to shared memory
      
     s_input[ty] = d_Buffer[threadIdx.y*imageW + blockId*blockDim.x + threadIdx.x];
      __syncthreads(); // wait for all shared data to load
      
	// result
      for (k = -filterR; k <= filterR; k++) {
	int d = threadIdx.y + k;
	
	if (d >= 0 && (d < imageW)) {
          sum += s_input[ty + k] * d_Filter[filterR - k];
        }     

        d_Output_GPU[threadIdx.y*imageW + blockId*blockDim.x + threadIdx.x] = sum;	
      }
      
}
 
 
 
 
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(TYPE *h_Dst, TYPE *h_Src, TYPE *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      TYPE sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(TYPE *h_Dst, TYPE *h_Src, TYPE *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      TYPE sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    TYPE
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU,
    *d_Input,
    *d_Output_GPU,
    *d_Buffer;
    
    int imageW;
    int imageH;
    unsigned int i;

    //printf("Enter filter radius : ");
    //scanf("%d", &filter_radius);
    //filter_radius = 16;
	
    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

 //   printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
  //  printf("Allocating and initializing host arrays...\n");

    h_Filter    = (TYPE *)malloc(FILTER_LENGTH * sizeof(TYPE));
    h_Input     = (TYPE *)malloc(imageW * imageH * sizeof(TYPE));
    h_Buffer    = (TYPE *)malloc(imageW * imageH * sizeof(TYPE));
    h_OutputCPU = (TYPE *)malloc(imageW * imageH * sizeof(TYPE));
    h_OutputGPU = (TYPE *)malloc(imageW * imageH * sizeof(TYPE));     
    
    if( h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL ){
      printf("Malloc allocation problem on host, exiting...\n");
      return(1);
    }
      
    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (TYPE)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (TYPE)rand() / ((TYPE)RAND_MAX / 255) + (TYPE)rand() / (TYPE)RAND_MAX;
    }
// comment out CPU code since it is correct
/*
    printf("CPU computation...\n");

    clock_t begin = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); 
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); 
    clock_t end = clock();
    double cpu_time = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Cpu time: %lf\n",cpu_time);
    */
    hipMalloc((void**)&d_Input, imageW * imageH * sizeof(TYPE));
    hipMalloc((void**)&d_Output_GPU,  imageW * imageH * sizeof(TYPE));
    hipMalloc((void**)&d_Buffer,  imageW * imageH * sizeof(TYPE));

    if(!(d_Input || d_Output_GPU || d_Buffer)){
	printf("Malloc allocation problem on device, exiting.. \n");
	return(1);
    }
    
    
    //TIME 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float tot_time=0, timer = 0;
    
    
    
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(TYPE), hipMemcpyHostToDevice);
        // d_Filter is in __constant__ memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter, sizeof(TYPE) * FILTER_LENGTH);
    
    cudaCheckError(); 
    
    //block & grid dimensions
    int x_block, y_block, x_grid, y_grid;

    
    if(imageW < 1024){
	x_block = imageW;
	y_block = 1024 / imageW;
	
	x_grid = 1;
	y_grid = (imageW*imageW)/1024; 
    } 
    else{
      if( imageW == 1024){
	x_block = 1024;
	y_block = 1;
	
	x_grid = 1;
	y_grid = imageW;
      }
      else{
	x_block = 1024;
	y_block = 1;
	
	x_grid = (imageW*imageW)/1024;
	y_grid = imageH;
      }
    }
     
    dim3 grid(x_grid , y_grid);  
    dim3 block(x_block, y_block);

    
    hipEventRecord(start);
    convolROWS<<<grid , block>>>(d_Buffer, d_Input, imageH, imageW, filter_radius);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timer, start, stop);
    printf("Kernel time for rows: %f\n", timer);
    tot_time += timer;
    
    //wait 1st kernel to finish
    hipDeviceSynchronize();
    //check for errors
    cudaCheckError();
    

    dim3 grid2(y_grid , x_grid);  
    dim3 block2(y_block, x_block);
    
    hipEventRecord(start);
    convolCOL<<<grid2 , block2>>>(d_Output_GPU, d_Buffer, imageH, imageW, filter_radius);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timer, start, stop);
    printf("Kernel time for col: %f\n", timer);
    tot_time += timer;   
    

    //wait to finish
    hipDeviceSynchronize();
    //check for errors
    cudaCheckError();
    
    //copy output from device to host
    hipMemcpy(h_OutputGPU, d_Output_GPU, imageW * imageH * sizeof(TYPE), hipMemcpyDeviceToHost);
    
    printf("Total time for GPU: %f\n", tot_time);
    
    // comment out CPU GPC comparison code since it is correct
   
    //compare 
  /*  
    TYPE max = 0.0;
    unsigned int counter=0;
    
     for (i = 0; i < imageW * imageH; i++) {
	if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) > max){
	  max = ABS(h_OutputGPU[i] - h_OutputCPU[i]);

	  printf("iteration= %d max=%10g \n",i, max);
	  counter++;
	}
	
     }
     printf("for %d filter, max= %d, counter=%d\n", filter_radius, max, counter);
     
     */
     
    
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_OutputGPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    hipFree(d_Input);
    hipFree(d_Output_GPU);

    hipDeviceReset();


    return 0;
}
